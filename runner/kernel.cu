#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream> // Standard C++ library for console I/O
#include <string> // Standard C++ Library for string manip

#include <Windows.h> // WinAPI Header
#include <TlHelp32.h> //WinAPI Process API
#include <string>
#include <stdio.h>
#include <vector>
#include <iostream>

#include "payload.h"
#include "libpeconv-master/libpeconv/include/peconv.h" // include libPeConv header

void cudaStatus(hipError_t status)
{
    if (status != hipSuccess)
    {
        std::cout << "cuda call failed with: " << hipGetErrorString(status) << std::endl;
        std::exit(-1);
    }
}
// decode input data and write to output, each thread will decode 1 byte of output data
__global__ void decode(const char* input, char* output)
{
	// each thread will decode 1 byte of output data
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// each thread will decode 2 bytes of input data
	int input_index = index * 2;
	
	// initialize a variable to store the decoded byte
	char decoded = 0;
	// decode 1st nibble
	if (input[input_index] >= '0' && input[input_index] <= '9')
	{
		decoded = (input[input_index] - '0') << 4;
	}
	else if (input[input_index] >= 'a' && input[input_index] <= 'f')
	{
		decoded = (input[input_index] - 'a' + 10) << 4;
	}
	else if (input[input_index] >= 'A' && input[input_index] <= 'F')
	{
		decoded = (input[input_index] - 'A' + 10) << 4;
	}
	// decode 2nd nibble
	if (input[input_index + 1] >= '0' && input[input_index + 1] <= '9')
	{
		decoded |= (input[input_index + 1] - '0');
	}
	else if (input[input_index + 1] >= 'a' && input[input_index + 1] <= 'f')
	{
		decoded |= (input[input_index + 1] - 'a' + 10);
	}
	else if (input[input_index + 1] >= 'A' && input[input_index + 1] <= 'F')
	{
		decoded |= (input[input_index + 1] - 'A' + 10);
	}
	// write decoded byte to output
	output[index] = decoded;
}

// program obfuscation using GPU resources test application
// the objective of this sample is to hide program logic from reverse engineering tools
// by placing the logic in gpu memory and streaming it to the host in small blocks
// the host side of the program will interpret the instructions and execute them
// for this basic test, the program will load hex encoded binary to the gpu and decode it
// the decoded data will then be read back and executed by the host

// uses the "run portable executable from memory technique" to hide the program logic
// https://github.com/codecrack3/Run-PE---Run-Portable-Executable-From-Memory/blob/master/RunPE.cpp

// another approach is to store the program as llvm bitcode, an use use the llvm interpreter to execute it

// use peConv to load the paylod array into an executable and run it
int RunPortableExecutable(BYTE* payload, int payloadSize)
{
	//load the payload as a PE module:
	size_t size = 0;

	// load the DLL, function is a bit misnamed
	BYTE* pe_module = peconv::load_pe_executable(payload, payloadSize, size);

	if (!pe_module) {
		std::cout << "Failed loading PE" << std::endl;
		return -1;
	}
	//find the exported function in the payload
	FARPROC runtimeLoadedFunction_Pos = peconv::get_exported_func(pe_module, "runtimeLoadedFunction");
	if (!runtimeLoadedFunction_Pos) {
		std::cout << "Failed to find runtimeLoadedFunction" << std::endl;
		return -1;
	}
	//cast the found function to the type that it is supposed to have (this is required for the correct call)
	runtimeLoadedFunction = (void (_cdecl *) (const char*, const char*)) runtimeLoadedFunction_Pos;

	//prepare the string that will be passed to the imported function
	const char testString[] = "Hello EC700_A1 from GPU imported function!";

	//call the imported function
	runtimeLoadedFunction(testString, "hello_world.txt");

	//clean up
	memset(pe_module, 0, payloadSize);
	peconv::free_pe_buffer(pe_module, payloadSize);
	return 0;
}

int main()
{

	//// allocate gpu memory for encoded instructions
    char* gpu_encoded_instructions = nullptr;
	const size_t payload_size = strlen(encoded_payload);
	const size_t decoded_size = payload_size / 2;
	cudaStatus(hipMalloc(&gpu_encoded_instructions, payload_size));
    cudaStatus(hipMemcpy(gpu_encoded_instructions, encoded_payload, payload_size, hipMemcpyHostToDevice));
    
    //allocate gpu memory for decoded instructions
    char* gpu_decoded_instructions = nullptr;
    cudaStatus(hipMalloc(&gpu_decoded_instructions, decoded_size));
	
	// launch gpu kernel to decode the payload, each thread will decode 1 byte of output data
    decode<<<decoded_size, 1 >>>(gpu_encoded_instructions, gpu_decoded_instructions);

	// check for device side errors
	cudaStatus(hipGetLastError());

	// synchronize host and device
    cudaStatus(hipDeviceSynchronize());

	// allocate space on host for decoded instructions
    BYTE* executionBuffer = new BYTE[decoded_size];
	// copy decoded instructions from gpu to host
    cudaStatus(hipMemcpy(executionBuffer, gpu_decoded_instructions, decoded_size, hipMemcpyDeviceToHost));

	// overwrite decoded gpu memory region with 0s immediately after reading it
	cudaStatus(hipMemset(gpu_decoded_instructions, 0, decoded_size));
    
	// execute instructions
	int status = RunPortableExecutable(executionBuffer, decoded_size);

	// overwrite execution buffer with 0s immediately after executing it
	memset(executionBuffer, 0, decoded_size);
	delete(executionBuffer);

	// clear encoded gpu memory region before freeing it
	cudaStatus(hipMemset(gpu_encoded_instructions, 0, payload_size));

	
	cudaStatus(hipFree(gpu_encoded_instructions));
	cudaStatus(hipFree(gpu_decoded_instructions));
	return status;
}